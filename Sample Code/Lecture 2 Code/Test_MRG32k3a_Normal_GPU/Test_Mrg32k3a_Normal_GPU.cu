#include "hip/hip_runtime.h"
// Test_MRG32k3a_Normal_GPU.cu 
//
// Use hiprand to simulate the normal random variables
// (1) using hiprand normal and(2) hiprand uniformwith normal inverse function
// use MRG32k3a
//
//

#include "hip/hip_runtime.h"
#include ""

#include "stdafx.h"
#include "Test_MRG32k3a_Normal_kernel.cuh"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

//    These are required for Numerical Recipes in C

using namespace std;

/* Global Variables */
//	These variables are used by MRG32k3a
#define norm 2.328306549295728e-10
#define m1   4294967087.0
#define m2   4294944443.0
#define a12     1403580.0
#define a13n     810728.0
#define a21      527612.0
#define a23n    1370589.0

#define NR_END 1
#define FREE_ARG char*

void RollSeed_MRG32k3a(double *dseed);
void SkipAhead_MRG32k3a(int n, unsigned int **An1, unsigned int **An2);
unsigned int **uint_matrix(int nrl, int nrh, int ncl, int nch);
void free_uint_matrix(unsigned int **m, int nrl, int nrh, int ncl, int nch);

const unsigned int im1 = 4294967087;
const unsigned int im2 = 4294944443;
const unsigned int ia12 = 1403580;
const unsigned int ia13n = 810728;
const unsigned int ia21 = 527612;
const unsigned int ia23n = 1370589;

FILE *fout;

int main()
{
	int i, j, n, n1, n2, cudaNumber, ind, nSim, ii, nTimeStepsPerYear;
	unsigned int **An1, **An2, *seed, *h_seeds;
	unsigned long long lp1, lp2, seed1[3], seed2[3], sseed1[3], sseed2[3];
	double x, x2, S0, *dseed;
	unsigned int s11, s12, s13, s21, s22, s23, ib1, ib2, ia_k2, ia_k1;
	float r, sigma, dt, log_S0, *h_MRG_rng1, *h_MRG_rng2, *h_MRG_rng3;
	float *d_MRG_rng;
	unsigned int *d_seeds;
	hiprandStateMRG32k3a *devMRGStates;

	double time0, time1, time2, time3, time4, time5, time6;
	struct _timeb timebuffer;
	errno_t errcheck;
	hipError_t cudaStatus;

	errcheck = fopen_s(&fout, "Test_MRG32k3a_Normal_GPU.txt", "w");
	if (errcheck) printf(" Test_MRG32k3a_Normal_GPU.txt not opened \n");

	fprintf(fout, " Test MRG32k3a normal random number generation on GPU \n");

	//	Read inputs from text file
	fprintf(fout, " Now reading input files \n");
	FILE *fin;
	errcheck = fopen_s(&fin, "Test_MRG32k3a_GPU_Parameters.txt", "r");
	if (errcheck) printf(" File Test_MRG32k3a_GPU_Parameters.txt not opened \n");

	fscanf_s(fin, " %i %i %i  %i ", &ind, &n, &nSim, &i);
	fscanf_s(fin, " %f %f %lf %i ", &r, &sigma, &S0, &nTimeStepsPerYear);

	dt = 1.0 / nTimeStepsPerYear;
	x = log(S0);
	log_S0 = x;
	if (i > 0) s11 = i;
	else s11 = -i;

	printf("  Inputs: n, simulations per path %i  nSim %i \n", n, nSim);
	printf("  r = %f  sigma = %f   S0 = %f   nTimeStepsPerYear = %i  \n", r, sigma, S0, nTimeStepsPerYear);

	An1 = uint_matrix(0, 2, 0, 2);
	An2 = uint_matrix(0, 2, 0, 2);
	seed = (unsigned int *)malloc(6 * sizeof(unsigned int));
	dseed = (double *)malloc(6 * sizeof(double));

//	Initial seeds for this test run
//	s11 = 298193;  
	s12 = 104959;  s13 = 84736;
	s21 = 727366;  s22 = 94727;   s23 = 5928384;

	dseed[0] = s11;
	dseed[1] = s12;
	dseed[2] = s13;
	dseed[3] = s21;
	dseed[4] = s22;
	dseed[5] = s23;

	seed[0] = s11;
	seed[1] = s12;
	seed[2] = s13;
	seed[3] = s21;
	seed[4] = s22;
	seed[5] = s23;

	printf("  initial seeds =  %u %u %u %u %u %u \n", s11, s12, s13, s21, s22, s23);

	printf(" \n  Now running time test on GPU, n = %i simulations per path and nSim = %i separate simuation paths \n",
				n, nSim);

	_ftime64_s(&timebuffer);
	time0 = timebuffer.time + timebuffer.millitm / 1000.0;

	cudaNumber = 0;
	cudaStatus = hipSetDevice(cudaNumber);

	h_seeds = (unsigned int *)malloc((nSim*6) * sizeof(unsigned int));
	h_MRG_rng1 = (float *)malloc(nSim*sizeof(float));
	h_MRG_rng2 = (float *)malloc(nSim*sizeof(float));
	h_MRG_rng3 = (float *)malloc(nSim * sizeof(float));

	hipMalloc((void **)&d_seeds, (nSim*6)*sizeof(unsigned int));
	hipMalloc((void **)&d_MRG_rng, nSim*sizeof(float));

	_ftime64_s(&timebuffer);
	time1 = timebuffer.time + timebuffer.millitm / 1000.0;

		h_seeds[0] = s11;
		h_seeds[1] = s12;
		h_seeds[2] = s13;
		h_seeds[3] = s21;
		h_seeds[4] = s22;
		h_seeds[5] = s23;
		for (ii = 1; ii < nSim; ii++) {
			for (i = 0; i < 3; i++) {
				seed1[i] = h_seeds[(ii-1)*6 + i];
				seed2[i] = h_seeds[(ii-1)*6 + i + 3];
			}
			for (i = 0; i < 3; i++) {
				sseed1[i] = 0.0;
				sseed2[i] = 0.0;
				for (j = 0; j < 3; j++) {
					sseed1[i] += (An1[i][j] * seed1[j]) % im1;
					sseed2[i] += (An2[i][j] * seed2[j]) % im2;
				}
				lp1 = sseed1[i];
				lp1 = lp1 % im1;
				if (lp1 < 0) lp1 += im1;
				h_seeds[ii*6+i] = lp1;
				lp2 = sseed2[i];
				lp2 = lp2 % im2;
				if (lp2 < 0) lp2 += im2;
				h_seeds[ii*6+i+3] = lp2;
			}
		}

	hipMemcpy(d_seeds, h_seeds, nSim*6*sizeof(unsigned int), hipMemcpyHostToDevice);

	hipMalloc((void **)&devMRGStates, nSim*sizeof(hiprandStateMRG32k3a));

	_ftime64_s(&timebuffer);
	time0 = timebuffer.time + timebuffer.millitm / 1000.0;

	setup_kernel << <(1 + nSim / 64), 64 >> >(nSim, devMRGStates);

	_ftime64_s(&timebuffer);
	time1 = timebuffer.time + timebuffer.millitm / 1000.0;

	Test_curand_MRG32k3a <<<(1 + nSim / 64), 64 >>>(
		n, nSim, r, sigma, dt, log_S0, d_seeds, d_MRG_rng, devMRGStates);

	hipGetLastError();
	hipDeviceSynchronize();

	hipMemcpy(h_MRG_rng1, d_MRG_rng, nSim*sizeof(float), hipMemcpyDeviceToHost);

	_ftime64_s(&timebuffer);
	time2 = timebuffer.time + timebuffer.millitm / 1000.0;

	Test_curand_MRG32k3a_NormInv <<<(1 + nSim / 64), 64 >>>(
		n, nSim, r, sigma, dt, log_S0, d_seeds, d_MRG_rng, devMRGStates);

	hipGetLastError();
	hipDeviceSynchronize();

	hipMemcpy(h_MRG_rng2, d_MRG_rng, nSim*sizeof(float), hipMemcpyDeviceToHost);

	_ftime64_s(&timebuffer);
	time3 = timebuffer.time + timebuffer.millitm / 1000.0;


	Test_MRG32k3a_NormInv << <(1 + nSim / 64), 64 >> > (
		n, nSim, r, sigma, dt, log_S0, d_seeds, d_MRG_rng);

	hipGetLastError();
	hipDeviceSynchronize();

	hipMemcpy(h_MRG_rng3, d_MRG_rng, nSim * sizeof(float), hipMemcpyDeviceToHost);

	_ftime64_s(&timebuffer);
	time4 = timebuffer.time + timebuffer.millitm / 1000.0;

	time4 = time4 - time3;
	time3 = time3 - time2;
	time2 = time2 - time1;
	time1 = time1 - time0;

	printf("  Run times: MRG hiprand_normal %7.3lf  MRG hiprand_uniform with norminv %7.3lf   for setup  %7.3lf \n", time2, time3, time1);
	printf("  Run times: MRG uniform with norminv %7.3lf \n", time4);

	printf("  Print first 20 rows and last 2 rows of the MRG32k3a simulations on GPU \n");
	printf("    MRG(hiprand)      MRG(hiprand-norminv)    MRG(norm-inv) \n");
	for (i = 0; i < 20; i++) printf("   %12.8f        %12.8f        %12.8f \n", 
					h_MRG_rng1[i], h_MRG_rng2[i], h_MRG_rng3[i]);
	printf("\n   %12.8f        %12.8f        %12.8f \n", h_MRG_rng1[nSim - 2], h_MRG_rng2[nSim - 2], h_MRG_rng3[nSim - 2]);
	printf("\n   %12.8f        %12.8f        %12.8f \n \n", h_MRG_rng1[nSim - 1], h_MRG_rng2[nSim - 1], h_MRG_rng3[nSim - 1]);

	_ftime64_s(&timebuffer);
	time4 = timebuffer.time + timebuffer.millitm / 1000.0;

	
	free(seed);
	free(dseed);
	free_uint_matrix(An1, 0, 2, 0, 2);
	free_uint_matrix(An2, 0, 2, 0, 2);

	free(h_seeds);
	free(h_MRG_rng1);
	free(h_MRG_rng2);
	free(h_MRG_rng3);

	hipFree(d_seeds);
	hipFree(d_MRG_rng);

	hipDeviceReset();

	return 0;

}

void RollSeed_MRG32k3a(double *dseed)
{
	int k;
	double p1, p2;

	p1 = a12*dseed[1] - a13n*dseed[2];
	k = p1 / m1;
	p1 -= k*m1;
	if (p1 < 0.0) p1 += m1;
	dseed[2] = dseed[1]; dseed[1] = dseed[0]; dseed[0] = p1;

	p2 = a21*dseed[3] - a23n*dseed[5];
	k = p2 / m2;
	p2 -= k*m2;
	if (p2 < 0.0) p2 += m2;
	dseed[5] = dseed[4]; dseed[4] = dseed[3]; dseed[3] = p2;

	return;

}

//	This method is slightly faster and more accurate (always spot on, matches brute force rolling of seeds)
//	This method is very fast
void SkipAhead_MRG32k3a(int n, unsigned int **An1, unsigned int **An2)
{
	int i, j, ii;
	long long kmod, lp1, lp2;
	long long A1[3][3], A2[3][3], B1[3][3], B2[3][3], C1[3][3], C2[3][3];

	A1[0][0] = 0; A1[0][1] = ia12;
	A1[0][2] = 0;
	A1[0][2] -= ia13n;
	//	A1[0][2] = -ia13n;
	A1[1][0] = 1; A1[1][1] = 0; A1[1][2] = 0;
	A1[2][0] = 0; A1[2][1] = 1; A1[2][2] = 0;

	A2[0][0] = ia21; A2[0][1] = 0;
	A2[0][2] = 0;
	A2[0][2] -= ia23n;
	//	A2[0][2] = -ia23n;
	A2[1][0] = 1; A2[1][1] = 0; A2[1][2] = 0;
	A2[2][0] = 0; A2[2][1] = 1; A2[2][2] = 0;

	for (i = 0; i < 3; i++) {
		for (j = 0; j < 3; j++) {
			B1[i][j] = A1[i][j];
			B2[i][j] = A2[i][j];
		}
	}

	for (ii = 1; ii <= (n - 1); ii++) {
		//	pre-multiply by Ai, calculating with 64 bit signed integers
		C1[0][0] = A1[0][0] * B1[0][0] + A1[0][1] * B1[1][0] + A1[0][2] * B1[2][0];
		C1[0][1] = A1[0][0] * B1[0][1] + A1[0][1] * B1[1][1] + A1[0][2] * B1[2][1];
		C1[0][2] = A1[0][0] * B1[0][2] + A1[0][1] * B1[1][2] + A1[0][2] * B1[2][2];
		C1[1][0] = A1[1][0] * B1[0][0] + A1[1][1] * B1[1][0] + A1[1][2] * B1[2][0];
		C1[1][1] = A1[1][0] * B1[0][1] + A1[1][1] * B1[1][1] + A1[1][2] * B1[2][1];
		C1[1][2] = A1[1][0] * B1[0][2] + A1[1][1] * B1[1][2] + A1[1][2] * B1[2][2];
		C1[2][0] = A1[2][0] * B1[0][0] + A1[2][1] * B1[1][0] + A1[2][2] * B1[2][0];
		C1[2][1] = A1[2][0] * B1[0][1] + A1[2][1] * B1[1][1] + A1[2][2] * B1[2][1];
		C1[2][2] = A1[2][0] * B1[0][2] + A1[2][1] * B1[1][2] + A1[2][2] * B1[2][2];

		C2[0][0] = A2[0][0] * B2[0][0] + A2[0][1] * B2[1][0] + A2[0][2] * B2[2][0];
		C2[0][1] = A2[0][0] * B2[0][1] + A2[0][1] * B2[1][1] + A2[0][2] * B2[2][1];
		C2[0][2] = A2[0][0] * B2[0][2] + A2[0][1] * B2[1][2] + A2[0][2] * B2[2][2];
		C2[1][0] = A2[1][0] * B2[0][0] + A2[1][1] * B2[1][0] + A2[1][2] * B2[2][0];
		C2[1][1] = A2[1][0] * B2[0][1] + A2[1][1] * B2[1][1] + A2[1][2] * B2[2][1];
		C2[1][2] = A2[1][0] * B2[0][2] + A2[1][1] * B2[1][2] + A2[1][2] * B2[2][2];
		C2[2][0] = A2[2][0] * B2[0][0] + A2[2][1] * B2[1][0] + A2[2][2] * B2[2][0];
		C2[2][1] = A2[2][0] * B2[0][1] + A2[2][1] * B2[1][1] + A2[2][2] * B2[2][1];
		C2[2][2] = A2[2][0] * B2[0][2] + A2[2][1] * B2[1][2] + A2[2][2] * B2[2][2];

		for (i = 0; i < 3; i++) {
			for (j = 0; j < 3; j++) {
				lp1 = C1[i][j];
				lp1 = lp1 % im1;
				if (lp1 < 0) lp1 += im1;
				B1[i][j] = lp1;
				lp2 = C2[i][j];
				lp2 = lp2 % im2;
				if (lp2 < 0) lp2 += im2;
				B2[i][j] = lp2;
			}
		}
	}

	for (i = 0; i < 3; i++) {
		for (j = 0; j < 3; j++) {
			An1[i][j] = B1[i][j];
			An2[i][j] = B2[i][j];
		}
	}

	return;

}

unsigned int **uint_matrix(int nrl, int nrh, int ncl, int nch)
/* allocate an unsigned int matrix with subscript range m[nrl..nrh][ncl..nch] */
{
	int i, nrow = nrh - nrl + 1, ncol = nch - ncl + 1;
	unsigned int **m;

	/* allocate pointers to rows */
	m = (unsigned int **)malloc((size_t)((nrow + NR_END) * sizeof(unsigned int*)));
	m += NR_END;
	m -= nrl;

	/* allocate rows and set pointers to them */
	m[nrl] = (unsigned int *)malloc((size_t)((nrow*ncol + NR_END) * sizeof(unsigned int)));
	m[nrl] += NR_END;
	m[nrl] -= ncl;

	for (i = nrl + 1; i <= nrh; i++) m[i] = m[i - 1] + ncol;

	/* return pointer to array of pointers to rows */
	return m;
}

void free_uint_matrix(unsigned int **m, int nrl, int nrh, int ncl, int nch)
/* free a double matrix allocated by dmatrix() */
{
	free((FREE_ARG)(m[nrl] + ncl - NR_END));
	free((FREE_ARG)(m + nrl - NR_END));
}


#undef norm
#undef m1
#undef m2
#undef a12
#undef a13n
#undef a21
#undef a23n
#undef NR_END
#undef FREE_ARG

//#undef MAX_NUM_THREADS

/* (C)Copr. 1986-92 Numerical Recipes Software G2v#X):K. */

